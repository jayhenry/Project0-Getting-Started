#include "hip/hip_runtime.h"
#include "common.h"

#include <>

#include <cmath>
#include <iostream>
#include <random>

// TODO 10: Implement the matrix multiplication kernel
__global__ void matrixMultiplicationNaive(float* const matrixP, const float* const matrixM, const float* const matrixN,
                                          const unsigned sizeMX, const unsigned sizeNY, const unsigned sizeXY)
{
    // TODO 10a: Compute the P matrix global index for each thread along x and y dimentions.
    // Remember that each thread of the kernel computes the result of 1 unique element of P
    unsigned px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned py = blockIdx.y * blockDim.y + threadIdx.y;

    // TODO 10b: Check if px or py are out of bounds. If they are, return.
    if (px >= sizeMX || py >= sizeNY)
		return;

    // TODO 10c: Compute the dot product for the P element in each thread
    // This loop will be the same as the host loop
    float dot = 0.0;
    for (unsigned k = 0; k < sizeXY; ++k) {
        dot += matrixM[k * sizeMX + px] * matrixN[py * sizeXY + k];
	}

    // TODO 10d: Copy dot to P matrix
	matrixP[py * sizeMX + px] = dot;
}

int main(int argc, char *argv[])
{
    // TODO 1: Initialize sizes. Start with simple like 16x16, then try 32x32.
    // Then try large multiple-block square matrix like 64x64 up to 2048x2048.
    // Then try square, non-power-of-two like 15x15, 33x33, 67x67, 123x123, and 771x771
    // Then try rectangles with powers of two and then non-power-of-two.
    const unsigned sizeMX = 46;
    const unsigned sizeXY = 123;
    const unsigned sizeNY = 771;

    // TODO 2: Allocate host 1D arrays for:
    // matrixM[sizeMX, sizeXY]
    // matrixN[sizeXY, sizeNY]
    // matrixP[sizeMX, sizeNY]
    // matrixPGold[sizeMX, sizeNY]
    float* matrixM = new float[sizeMX * sizeXY];
    float* matrixN = new float[sizeXY * sizeNY];
    float* matrixP = new float[sizeMX * sizeNY];
    float* matrixPGold = new float[sizeMX * sizeNY];

    // LOOK: Setup random number generator and fill host arrays and the scalar a.
    std::random_device rd;
    std::mt19937 mt(rd());
    std::uniform_real_distribution<float> dist(0.0, 1.0);

    // Fill matrix M on host
    for (unsigned i = 0; i < sizeMX * sizeXY; i++)
        matrixM[i] = dist(mt);

    // Fill matrix N on host
    for (unsigned i = 0; i < sizeXY * sizeNY; i++)
        matrixN[i] = dist(mt);

    std::cout << "***Compute gold on CPU***" << std::endl;
    // TODO 3: Compute "gold" reference standard
    // for py -> 0 to sizeNY
    //   for px -> 0 to sizeMX
    //     initialize dot product accumulator
    //     for k -> 0 to sizeXY
    //       dot = m[k, px] * n[py, k]
    //  matrixPGold[py, px] = dot
    for(int py=0; py < sizeNY; py++) {
        for(int px=0; px < sizeMX; px++) {
            float dot = 0.0;
            for(int k=0; k < sizeXY; k++) {
                dot += matrixM[k * sizeMX + px] * matrixN[py * sizeXY + k];
            }
            matrixPGold[py * sizeMX + px] = dot;
        }
	}

    // Device arrays
    float *d_matrixM, *d_matrixN, *d_matrixP;

    std::cout << "***Mem Alloc on GPU***" << std::endl;
    // TODO 4: Allocate memory on the device for d_matrixM, d_matrixN, d_matrixP.
	CUDA(hipMalloc((void**)&d_matrixM, sizeMX * sizeXY * sizeof(float)));
	CUDA(hipMalloc((void**)&d_matrixN, sizeNY * sizeXY * sizeof(float)));
	CUDA(hipMalloc((void**)&d_matrixP, sizeMX * sizeNY * sizeof(float)));

    std::cout << "***MemcpyHostToDevice***" << std::endl;
    // TODO 5: Copy array contents of M and N from the host (CPU) to the device (GPU)
	CUDA(hipMemcpy(d_matrixM, matrixM, sizeMX * sizeXY * sizeof(float), hipMemcpyHostToDevice));
	CUDA(hipMemcpy(d_matrixN, matrixN, sizeNY * sizeXY * sizeof(float), hipMemcpyHostToDevice));

    CUDA(hipDeviceSynchronize());

    ////////////////////////////////////////////////////////////
    std::cout << "****************************************************" << std::endl;
    std::cout << "***Matrix Multiplication***" << std::endl;

    // LOOK: Use the clearHostAndDeviceArray function to clear matrixP and d_matrixP
    clearHostAndDeviceArray(matrixP, d_matrixP, sizeMX * sizeNY);

    // TODO 6: Assign a 2D distribution of BS_X x BS_Y x 1 CUDA threads within
    // Calculate number of blocks along X and Y in a 2D CUDA "grid" using divup
    // HINT: The shape of matrices has no impact on launch configuaration
	const unsigned BS_X = 32;
	const unsigned BS_Y = 32;
	const unsigned numBlocksX = divup(sizeMX, BS_X);
	const unsigned numBlocksY = divup(sizeNY, BS_Y);
    DIMS dims;
    dims.dimBlock = dim3(BS_X, BS_Y, 1);
    dims.dimGrid  = dim3(numBlocksX, numBlocksY, 1);

    // TODO 7: Launch the matrix transpose kernel
     matrixMultiplicationNaive<<<dims.dimGrid, dims.dimBlock>>>(d_matrixP, d_matrixM, d_matrixN, sizeMX, sizeNY, sizeXY);

     hipError_t err = hipGetLastError();
     if (err != hipSuccess) {
         fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
         exit(1);
     }

     hipDeviceSynchronize();

    // TODO 8: copy the answer back to the host (CPU) from the device (GPU)
	 CUDA(hipMemcpy(matrixP, d_matrixP, sizeMX * sizeNY * sizeof(float), hipMemcpyDeviceToHost));

    // LOOK: Use compareReferenceAndResult to check the result
    compareReferenceAndResult(matrixPGold, matrixP, sizeMX * sizeNY, 1e-3);

    std::cout << "****************************************************" << std::endl << std::endl;
    ////////////////////////////////////////////////////////////

    // TODO 9: free device memory using hipFree
	CUDA(hipFree(d_matrixM));
	CUDA(hipFree(d_matrixN));
	CUDA(hipFree(d_matrixP));

    // free host memory
    delete[] matrixM;
    delete[] matrixN;
    delete[] matrixP;
    delete[] matrixPGold;

    // successful program termination
    return 0;
}
